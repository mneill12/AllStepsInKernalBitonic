#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <hip/hip_runtime_api.h>

#include "writeToCSVFileHeader.h"
#include "userInputHeader.h"

void printArray(int *elements);


int deviceBlocks;
int threadsPerBlock;
int elementsToSort;
int threadCount;

int phases;

//Max times we cann run the process
int executionCount;


const int randMax = 10000;

void createUnsortedArray(int* elements){

	for (int i = 0; i < elementsToSort; ++i){
		elements[i] = rand() % randMax - rand() % 5;
	}

}

bool isSorted(int *elements){

	bool sorted = true;
	for (int i = 0; i < (elementsToSort - 1); ++i){
		if (elements[i] > elements[i + 1]){
			sorted = false;
		}
	}
	return sorted;
}


double getElapsedTime(clock_t start, clock_t stop)
{
	double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
	printf("Elapsed time: %.3fs\n", elapsed);

	return elapsed;
}

int random_int()
{
	return (int)rand() / (int)2048;
}


__global__ void bitonicSort(int* deviceElements, int subSequenceSize, int steps){


	//1printf("Kernal Called!!!!");
	/*
	Here we get our first thread var i and j.
	we get j by knowing the size of the subsequence and then halfing it, this gives us the rang that values should be comapired for this step.
	As we go down the steps, we'll be halfing j until step = 1;
	*/

	int firstIndex = threadIdx.x + blockDim.x * blockIdx.x;
	int rangeOfComparison = (subSequenceSize / 2);
	for (int step = steps; step >= 1; step--){

		//This xor op checks that our second value is bigger than our firstIndex value
		if ((firstIndex ^ rangeOfComparison) > firstIndex){

			//assending
			if ((firstIndex / subSequenceSize) % 2 == 0){

				if (deviceElements[firstIndex] > deviceElements[firstIndex ^ rangeOfComparison]) {
					printf("Even element assending %d: %d -> %d \n", firstIndex, deviceElements[firstIndex], deviceElements[firstIndex ^ rangeOfComparison]);
					int temp = deviceElements[firstIndex];
					deviceElements[firstIndex] = deviceElements[firstIndex ^ rangeOfComparison];
					deviceElements[firstIndex ^ rangeOfComparison] = temp;
				}

			}
			else{

				if (deviceElements[firstIndex] < deviceElements[firstIndex ^ rangeOfComparison]) {
					printf("Even element desending %d : %d -> %d \n", firstIndex, deviceElements[firstIndex], deviceElements[firstIndex ^ rangeOfComparison]);
					int temp = deviceElements[firstIndex];
					deviceElements[firstIndex] = deviceElements[firstIndex ^ rangeOfComparison];
					deviceElements[firstIndex ^ rangeOfComparison] = temp;
				}

			}

		}
		__syncthreads();

		rangeOfComparison = rangeOfComparison / 2;
	}
}
/*
Main function call. Created array and calls stepskernel based of the size of the bitonic sequences and step.
*/
void bitonic_Sort(int* elements){

	int* d_elements;

	//get "phases" so we know how many times we need to send array over to device  
	phases = int(log2(double(elementsToSort)));

	//General cuda managment here : Allocate on device, array isn't going to change  in size
	hipMalloc(&d_elements, elementsToSort*sizeof(int));
	hipMemcpy(d_elements, elements, elementsToSort*sizeof(int), hipMemcpyHostToDevice);
	dim3 blocks(deviceBlocks, 1);    /* Number of blocks   */
	dim3 threads(threadsPerBlock, 1);  /* Number of threads  */

	for (int currentPhase = 1; currentPhase <= phases; currentPhase++){

		//Get the  size of each sub sequence and the amount of "Steps" in the individual sub sequences 
		int subSequenceSize = int(pow(double(2), double(currentPhase)));

		int steps = int(log2((double)subSequenceSize));

		hipDeviceSynchronize();
		printf("Phase: %d \n", currentPhase);
		hipDeviceSynchronize();
		bitonicSort << <blocks, threads>> >(d_elements, subSequenceSize, steps);
		hipDeviceSynchronize();
	}
	hipMemcpy(elements, d_elements, elementsToSort*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_elements);
}


void preExecution(){

	int values[7];
	values[0] = 10;
	values[1] = 13;
	values[2] = 9;
	values[3] = 18;
	values[4] = 26;
	values[4] = 100;
	values[6] = 3;

	bitonic_Sort(values);
}

int main(void)
{
	executionCount = getMaxProcessCount();
	int fixedExecutionCount = executionCount;

	preExecution();

	bool runSort = true;

	//Pointers to store our results that we're writing to CSV files, allocate space entered buy the user
	int* threadCounts = (int*)malloc(executionCount*sizeof(int));
	int* allBlocks = (int*)malloc(executionCount*sizeof(int));;
	double* timeResults = (double*)malloc(executionCount*sizeof(double));;
	char* arrayStates = (char*)malloc(executionCount*sizeof(char));

	double time;
	clock_t start, stop;
	//Counter so we can assine values to the array in the execution loop

	while (runSort && executionCount != 0){

		runSort = runSortAgain();

		//Get thread, blocks and  element count

		//Get total elements and suggested block thread configurations
		blockAndThreadCounts inputCountandSuggestedThreadBlockCount;
		inputCountandSuggestedThreadBlockCount = getElementCounts();
		elementsToSort = inputCountandSuggestedThreadBlockCount.elementCount;

		//wirte possible thread and block configurations to text file
		printf("Writing suggested block thread configuration...");
		writeSuggestedBlockThreadConfigToCsv(inputCountandSuggestedThreadBlockCount.threadCounts,
			inputCountandSuggestedThreadBlockCount.blockCounts,
			inputCountandSuggestedThreadBlockCount.combinationsCount
			);
		printf("Done \n");
		//elementsToSort = inputCountandSuggestedThreadBlockCount.elementCount;
		deviceBlocks = getBlockCount();
		threadsPerBlock = getThreadCount();

		threadCount = threadsPerBlock * deviceBlocks;

		//Malloc array, add values to it and write unsorted array to csv file
		int* values = (int*)malloc(elementsToSort*sizeof(int));
		createUnsortedArray(values);
		writeBlockElementCsvFile(values, "preSorted", threadCount, deviceBlocks);

		//Do Sort and time it
		start = clock();
		bitonic_Sort(values);
		stop = clock();

		time = getElapsedTime(start, stop);

		char* arrayState;
		char arrayStateChar;

		if (isSorted(values)){

			printf("Is Sorted \n");
			arrayState = "sorted";
			arrayStateChar = 's';
		}
		else{

			printf("Not Sorted \n");
			arrayState = "unsorted";
			arrayStateChar = 'u';
		}

		writeBlockElementCsvFile(values, arrayState, threadCount, deviceBlocks);

		//Allocate results values to pointers 
		*threadCounts = threadCount;
		*allBlocks = deviceBlocks;
		*timeResults = time;
		*arrayStates = arrayStateChar;

		//Increment Result pointers
		threadCounts++;
		allBlocks++;
		timeResults++;
		arrayStates++;

		free(values);

		//Check again for user input

		executionCount--;
	}

	printf("Execution ended. Writing results to C:\BitonicSortArrayCSVFiles /n");

	writeSortResultsToCsv(timeResults, "ParallelBitonicSort", arrayStates, threadCounts, allBlocks, fixedExecutionCount);

	getchar();
}
